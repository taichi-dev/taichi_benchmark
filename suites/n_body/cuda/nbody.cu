#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"

#define BLOCK_SIZE 256
#define SOFTENING 1e-9f

typedef struct { float x, y, z, vx, vy, vz; } Body;

void randomizeBodies(float *data, int n) {
  for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}

__global__
void bodyForce(Body *p, float dt, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

    for (int j = 0; j < n; j++) {
      float dx = p[j].x - p[i].x;
      float dy = p[j].y - p[i].y;
      float dz = p[j].z - p[i].z;
      float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      float invDist = rsqrtf(distSqr);
      float invDist3 = invDist * invDist * invDist;

      Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
    }

    p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
  }
}

int main(const int argc, const char** argv) {
  int nBodies = 30000;
  if (argc > 1) nBodies = atoi(argv[1]);
  
  const float dt = 0.01f; // time step
  const int nIters = 10;  // simulation iterations

  int bytes = nBodies * sizeof(Body);
  float *buf = (float*) malloc(bytes);
  Body *p = (Body*)buf;

  randomizeBodies(buf, 6 * nBodies); // Init pos / vel data

  float *d_buf;
  hipMalloc(&d_buf, bytes);
  Body *d_p = (Body*)d_buf;

  int nBlocks = (nBodies + BLOCK_SIZE - 1) / BLOCK_SIZE;
  double totalTime = 0.0; 

  for (int iter = 1; iter <= nIters; iter++) {
    Timer tmr;
    tmr.start();
    hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
    bodyForce<<<nBlocks, BLOCK_SIZE>>>(d_p, dt, nBodies); // compute interbody forces
    hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);

    for (int i = 0 ; i < nBodies; i++) { // integrate position
      p[i].x += p[i].vx*dt;
      p[i].y += p[i].vy*dt;
      p[i].z += p[i].vz*dt;
    }
    tmr.stop();
    const double tElapsed = tmr.getTimeMillisecond();
    if (iter > 1) { // First iter is warm up
      totalTime += tElapsed; 
    }
  }
  double avgTime = totalTime / (double)(nIters-1); 

#ifdef JSON_OUTPUT
  printf("{\"nbodies\":%d, \"rate\":%.3lf, \"time\": %.3lf}\n", nBodies, 1e-6 * nBodies * nBodies / avgTime, avgTime);
#else
  printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, 1e-6 * nBodies * nBodies / avgTime);
#endif
  free(buf);
  hipFree(d_buf);
}
